#include "hip/hip_runtime.h"
#include "pch.h"
#include <>
#include "Core.cuh"
#include "Interface.cuh"
#include "Matrix.cuh"
#include "Background.cuh"

#define RECURSION_DEPTH 5

namespace RayTracer
{
	enum class ColorType { REFLECTION, REFRACTION };

	RUN_ON_GPU_CALL_FROM_CPU void render(pixels pixels, const input* dinput, Projection proj_type);
	RUN_ON_GPU Core::vec3 cast_primary_ray(const world& models, ray& ray);
	RUN_ON_GPU Core::vec3 cast_second_ray(const ColorType type, const world& models, ray& ray);
	RUN_ON_GPU Core::vec3 get_reflect_dir(const Core::vec3& incident_dir, const Core::vec3& nhit);
	RUN_ON_GPU Core::vec3 get_refract_dir(const Core::vec3& incident_dir, const Core::vec3& nhit, const bool& inside);
	RUN_ON_GPU Core::vec3 cast_shadow_ray(const world& models, ray& ray, const hit& hit);
	RUN_ON_GPU model* get_camera(const world& models);
	RUN_ON_GPU void update_camera_position(const Core::mat4& translator, model* pcamera);
	RUN_ON_GPU double get_glow(const unsigned light_index, const world& models, const ray& shadow_ray);
}

void RayTracer::draw_frame(RayTracer::pixels pixels, input* dinput, Projection proj_type)
{
	dim3 block_size(32, 32, 1);
	dim3 grid_size(pixels.width / 32, pixels.height / 32, 1);
	render << < grid_size, block_size >> > (pixels, dinput, proj_type);
}

RUN_ON_GPU_CALL_FROM_CPU
void RayTracer::render(RayTracer::pixels pixels, const input* dinput, Projection proj_type)
{
	double aspect_ratio = pixels.width / pixels.height;
	double tan_val = tangent(dinput->fov/ 2.0);
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;
	double near_plane = (proj_type == Projection::PERSPECTIVE) ? 1.0 : 20.0;
	int index = (ty * pixels.width) + tx;
	double x = ((2.0 * ((tx + 0.5) / pixels.width)) - 1.0) * aspect_ratio * tan_val * near_plane;
	double y = (1.0 - (2.0 * ((ty + 0.5) / pixels.height))) * tan_val * near_plane;
	world* dworld = (world*)(dinput->dworld);
	model* pcamera = get_camera(*dworld);
	update_camera_position(dinput->translator, pcamera);
	Core::vec3 dir = (proj_type == Projection::PERSPECTIVE) ? Core::vec3{ x, y, -near_plane } : Core::vec3{ 0.0, 0.0, -near_plane };
	normalize(dir);
	Core::vec3 origin = (proj_type == Projection::PERSPECTIVE) ? Core::vec3{} : Core::vec3{ x, y };
	if(proj_type == Projection::PERSPECTIVE) dir = dinput->rotator * dir;
	origin = dinput->translator * origin;
	ray pray{ origin, dir };
	Core::vec3 color = cast_primary_ray(*dworld, pray);
	pixels.data[index] = rgb{ unsigned char(color.x * 255.0), unsigned char(color.y * 255.0), unsigned char(color.z * 255.0) };
}

RUN_ON_GPU
Core::vec3 RayTracer::cast_primary_ray(const world& models, ray& ray)
{
	Core::vec3 surface_color, background{ 1.0, 1.0, 1.0 };
	hit hit_item;
	if (!detect_hit(models, ray, hit_item)) return get_background_color(models.dcubemap, ray.dir);
	if (hit_item.pmodel->reflectivity > 0.0 || hit_item.pmodel->transparency > 0.0)
	{
		Core::vec3 reflection_color = (hit_item.pmodel->reflectivity > 0.0) ? cast_second_ray(ColorType::REFLECTION, models, ray) : background;
		Core::vec3 refraction_color = (hit_item.pmodel->transparency > 0.0) ? cast_second_ray(ColorType::REFRACTION, models, ray) : background;
		double fresnel = schlick_approximation(dot(-ray.dir, ray.nhit), 0.1);
		Core::vec3 diffuse_color = get_color(hit_item, ray, hit_item.pmodel->diffuse);
		surface_color = (reflection_color * fresnel + refraction_color * (1 - fresnel) * hit_item.pmodel->transparency) * diffuse_color;
	}
	else
	{
		surface_color = cast_shadow_ray(models, ray, hit_item);
	}
	return surface_color;
}

RUN_ON_GPU
Core::vec3 RayTracer::cast_second_ray(const ColorType type, const world& models, ray& pray)
{
	Core::vec3 color { 1.0, 1.0, 1.0 };
	double bias = 1e-4;
	bool inside = false;
	int depth = 0;
	hit hit_item;
	ray nray;
	nray.origin = (type == ColorType::REFRACTION) ? pray.phit - pray.nhit * bias : pray.phit;
	nray.dir = (type == ColorType::REFRACTION) ? get_refract_dir(pray.dir, pray.nhit, inside) : get_reflect_dir(pray.dir, pray.nhit);
	while ((depth < RECURSION_DEPTH) && detect_hit(models, nray, hit_item))
	{
		if ((type == ColorType::REFRACTION) ? (hit_item.pmodel->transparency > 0.0) : (hit_item.pmodel->reflectivity > 0.0))
		{
			color *= get_color(hit_item, nray, hit_item.pmodel->diffuse);
			nray.dir = (type == ColorType::REFRACTION) ? get_refract_dir(nray.dir, nray.nhit, inside) : get_reflect_dir(nray.dir, nray.nhit);
			nray.origin = (type == ColorType::REFRACTION) ? nray.phit - nray.nhit * bias : nray.phit;
			depth++;
		}
		else
		{
			color *= cast_shadow_ray(models, pray, hit_item);
			break;
		}
	}
	return color;
}

RUN_ON_GPU
Core::vec3 RayTracer::get_reflect_dir(const Core::vec3& incident_dir, const Core::vec3& nhit)
{
	Core::vec3 reflect_dir = incident_dir - (nhit * dot(incident_dir, nhit) * 2);
	normalize(reflect_dir);
	return reflect_dir;
}

RUN_ON_GPU
Core::vec3 RayTracer::get_refract_dir(const Core::vec3& incident_dir, const Core::vec3& nhit, const bool& inside)
{
	double ref_index_ratio = (inside) ? 1.1f : 1 / 1.1f;
	double cosine = dot(-incident_dir, nhit);
	Core::vec3 t1 = incident_dir * ref_index_ratio;
	Core::vec3 t2 = nhit * ((ref_index_ratio * cosine) - sqrt(1 - ((ref_index_ratio * ref_index_ratio) * (1 - (cosine * cosine)))));
	Core::vec3 refract_dir = t1 + t2;
	normalize(refract_dir);
	return refract_dir;
}

RUN_ON_GPU
Core::vec3 RayTracer::cast_shadow_ray(const world& models, ray& rray, const hit& hit)
{
	Core::vec3 color;
	Core::vec3 diffuse_color = get_color(hit, rray, hit.pmodel->diffuse);
	Core::vec3 specular_color = get_color(hit, rray, hit.pmodel->specular);
	Core::vec3 ambient_color = Core::vec3{ 0.25, 0.25, 0.25 };
	Core::vec3 ambient = diffuse_color * ambient_color;
	double bias = 1e-4;
	model* pcamera = get_camera(models);
	for (unsigned l = 0; l < models.size; l++)
	{
		model* light_model = &models.models[l];
		if ((light_model->m_type == Core::model_type::LIGHT) && (hit.pmodel->m_type == Core::model_type::OBJECT))
		{
			Core::vec3 shadow_dir = light_model->position - rray.phit;
			normalize(shadow_dir);
			Core::vec3 shadow_origin = rray.phit + rray.nhit * bias;
			ray shadow_ray{ shadow_origin, shadow_dir };
			double shadow_normal_dot = max_val(0.0, dot(rray.nhit, shadow_dir));
			Core::vec3 diffuse = diffuse_color * shadow_normal_dot;
			Core::vec3 reflect_dir = get_reflect_dir(-shadow_dir, rray.nhit);
			normalize(reflect_dir);
			Core::vec3 view_dir = pcamera->position - rray.phit;
			normalize(view_dir);
			Core::vec3 specular = specular_color * pow(max_val(0.0, dot(view_dir, reflect_dir)), 32) * shadow_normal_dot;
			color += (diffuse + specular) * get_glow(l, models, shadow_ray) * light_model->emissive_color;
		}
	}
	return get_clamped(color + hit.pmodel->emissive_color + ambient);
}

RUN_ON_GPU 
RayTracer::model* RayTracer::get_camera(const world& models)
{
	model* pcamera;
	for (unsigned i=0; i<models.size; i++)
	{
		model* pmodel = &models.models[i];
		if (pmodel->m_type == Core::model_type::CAMERA)
		{
			pcamera = pmodel;
			break;
		}
	}
	return pcamera;
}

RUN_ON_GPU 
void RayTracer::update_camera_position(const Core::mat4& translator, model* pcamera)
{
	pcamera->position = translator * Core::vec3{};
}

RUN_ON_GPU 
double RayTracer::get_glow(const unsigned light_index, const world& models, const ray& shadow_ray)
{
	double t0 = INFINITY, glow = 1.0;
	for (unsigned m = 0; m < models.size; m++)
	{
		if (m != light_index)
		{
			glow = get_glow_val(models.models[m], shadow_ray, t0);
			if (glow == 0.0) break;
		}
	}
	return glow;
}